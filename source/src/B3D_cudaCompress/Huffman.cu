#include "hip/hip_runtime.h"
#include <cudaCompress/Huffman.h>

#include <cassert>

#include <hip/hip_runtime.h>
//#include <thrust/device_ptr.h>
//#include <thrust/scan.h>

#include <cudaCompress/cudaUtil.h>
#include <cudaCompress/util.h>

#include <cudaCompress/InstanceImpl.h>


#include <cudaCompress/scan/scan_app.cui>

#include "HuffmanKernels.cui"


namespace cudaCompress {


size_t huffmanGetRequiredMemory(const Instance* pInstance)
{
    uint streamCountMax = pInstance->m_streamCountMax;
    uint symbolCountPerStreamMax = pInstance->m_elemCountPerStreamMax;

    size_t sizeEncode = 0;
    size_t sizeDecode = 0;

    // encode: dpStreamInfos
    sizeEncode += getAlignedSize(sizeof(HuffmanGPUStreamInfo) * streamCountMax, 128);

    // encode: dpScratch
    uint prefixCountMax = getPrefixCount(symbolCountPerStreamMax);
    uint scratchBytes = (uint)getAlignedSize((prefixCountMax + 1) * sizeof(uint), 128);
    sizeEncode += streamCountMax * getAlignedSize(scratchBytes, 128);
    // encode: dppScratch
    sizeEncode += getAlignedSize(streamCountMax * sizeof(uint*), 128);

    // encode: dpScanTotal
    sizeEncode += getAlignedSize(streamCountMax * sizeof(uint), 128);

    // decode: dpStreamInfos
    sizeDecode += getAlignedSize(sizeof(HuffmanGPUStreamInfo) * streamCountMax, 128);

    return max(sizeEncode, sizeDecode);
}

bool huffmanInit(Instance* pInstance)
{
    uint streamCountMax = pInstance->m_streamCountMax;

    cudaSafeCall(hipHostMalloc(&pInstance->Huffman.pReadback, streamCountMax * sizeof(uint)));

    cudaSafeCall(hipEventCreateWithFlags(&pInstance->Huffman.syncEventReadback, hipEventDisableTiming));

    return true;
}

bool huffmanShutdown(Instance* pInstance)
{
    cudaSafeCall(hipEventDestroy(pInstance->Huffman.syncEventReadback));
    pInstance->Huffman.syncEventReadback = 0;

    cudaSafeCall(hipHostFree(pInstance->Huffman.pReadback));
    pInstance->Huffman.pReadback = nullptr;

    return true;
}


bool huffmanEncode(Instance* pInstance, const HuffmanGPUStreamInfo* pStreamInfos, uint streamCount, uint codingBlockSize, uint* pCompressedSizeBits)
{
    assert(streamCount <= pInstance->m_streamCountMax);

    bool longSymbols = (pInstance->m_log2HuffmanDistinctSymbolCountMax > 16);

    uint prefixCountMax = 0;
    uint offsetCountMax = 0;
    for(uint i = 0; i < streamCount; i++) {
        const HuffmanGPUStreamInfo& streamInfo = pStreamInfos[i];

        assert(streamInfo.symbolCount <= pInstance->m_elemCountPerStreamMax);

        uint prefixCount = getPrefixCount(streamInfo.symbolCount);
        prefixCountMax = max(prefixCountMax, prefixCount);

        uint offsetCount = (streamInfo.symbolCount + codingBlockSize - 1) / codingBlockSize;
        offsetCountMax = max(offsetCountMax, offsetCount);
    }

    HuffmanGPUStreamInfo* dpStreamInfos = pInstance->getBuffer<HuffmanGPUStreamInfo>(streamCount);
    uint scratchElems = (uint)getAlignedSize(prefixCountMax + 1, 128 / sizeof(uint));
    uint* dpScratch = pInstance->getBuffer<uint>(streamCount * scratchElems);
    uint** dppScratch = pInstance->getBuffer<uint*>(streamCount);
    uint* dpScanTotal = pInstance->getBuffer<uint>(streamCount);

    std::vector<uint*> pdpScratch(streamCount);
    for(uint i = 0; i < streamCount; i++) {
        pdpScratch[i] = dpScratch + i * scratchElems;
    }


    util::CudaScopedTimer timer(pInstance->Huffman.timerEncode);

    timer("Upload Info");

    cudaSafeCall(hipMemcpyAsync(dpStreamInfos, pStreamInfos, sizeof(HuffmanGPUStreamInfo) * streamCount, hipMemcpyHostToDevice, pInstance->m_stream));
    // note: we don't sync on this upload - we trust that the caller won't overwrite/delete the array...

    cudaSafeCall(hipMemcpyAsync(dppScratch, pdpScratch.data(), sizeof(uint*) * streamCount, hipMemcpyHostToDevice, pInstance->m_stream)); //TODO upload buffer?
    // there's a sync in here later on, so this "should" be okay...

    timer("Words to Lengths");
    // get codeword lengths (of COMPACTIFY_ELEM_PER_THREAD consecutive codewords)
    if(prefixCountMax > 0) {
        uint blockSize = WORDS_TO_LENGTH_THREADS_PER_BLOCK;
        dim3 blockCount((prefixCountMax + blockSize - 1) / blockSize, streamCount);

        if(longSymbols) {
            huffmanEncodeWordsToLengthKernel<Symbol32><<<blockCount, blockSize, 0, pInstance->m_stream>>>(dppScratch, dpStreamInfos);
        } else {
            huffmanEncodeWordsToLengthKernel<Symbol16><<<blockCount, blockSize, 0, pInstance->m_stream>>>(dppScratch, dpStreamInfos);
        }
        cudaCheckMsg("huffmanEncodeWordsToLengthKernel execution failed");
    }

    timer("Scan Lengths");
    if(prefixCountMax > 0) {
        // scan codeword lengths to get output indices
        scanArray<uint, uint, true>(dpScratch, dpScratch, prefixCountMax + 1, streamCount, scratchElems, pInstance->m_pScanPlan, pInstance->m_stream);
        cudaCheckMsg("huffmanEncode: Error in scanArray");

        // copy scan totals (= compressed bit sizes) into contiguous buffer for common download
        uint blockSize = min(128u, streamCount);
        uint blockCount = (streamCount + blockSize - 1) / blockSize;
        huffmanEncodeCopyScanTotalsKernel<<<blockCount, blockSize, 0, pInstance->m_stream>>>(dpStreamInfos, streamCount, (const uint**)dppScratch, dpScanTotal);
        cudaCheckMsg("huffmanEncodeCopyScanTotalsKernel execution failed");

        // start readback of compressed size
        cudaSafeCall(hipMemcpyAsync(pInstance->Huffman.pReadback, dpScanTotal, streamCount * sizeof(uint), hipMemcpyDeviceToHost, pInstance->m_stream));
        cudaSafeCall(hipEventRecord(pInstance->Huffman.syncEventReadback, pInstance->m_stream));
    }

    timer("Collect Offsets");
    if(offsetCountMax > 0) {
        uint blockSize = min(128u, offsetCountMax);
        dim3 blockCount((offsetCountMax + blockSize - 1) / blockSize, streamCount);
        huffmanEncodeCollectOffsetsKernel<<<blockCount, blockSize, 0, pInstance->m_stream>>>(dpStreamInfos, (const uint**)dppScratch, codingBlockSize);
        cudaCheckMsg("huffmanEncodeCollectOffsetsKernel execution failed");
    }

    timer("Compactify");
    if(prefixCountMax > 0) {
        uint blockSize = COMPACTIFY_THREADS_PER_BLOCK;
        dim3 blockCount((prefixCountMax + blockSize - 1) / blockSize, streamCount);

        if(longSymbols) {
            huffmanEncodeCompactifyKernel<Symbol32><<<blockCount, blockSize, 0, pInstance->m_stream>>>(dpStreamInfos, (const uint**)dppScratch);
        } else {
            huffmanEncodeCompactifyKernel<Symbol16><<<blockCount, blockSize, 0, pInstance->m_stream>>>(dpStreamInfos, (const uint**)dppScratch);
        }
        cudaCheckMsg("huffmanEncodeCompactifyKernel execution failed");
    }

    timer("Readback Sync");

    if(prefixCountMax > 0) {
        cudaSafeCall(hipEventSynchronize(pInstance->Huffman.syncEventReadback));
    }
    for(uint i = 0; i < streamCount; i++) {
        const HuffmanGPUStreamInfo& streamInfo = pStreamInfos[i];

        if(streamInfo.symbolCount == 0) {
            pCompressedSizeBits[i] = 0;
        } else {
            pCompressedSizeBits[i] = pInstance->Huffman.pReadback[i];
        }
    }

    timer();

    pInstance->releaseBuffers(4);

    return true;
}

bool huffmanDecode(Instance* pInstance, const HuffmanGPUStreamInfo* pStreamInfos, uint streamCount, uint codingBlockSize)
{
    assert(streamCount <= pInstance->m_streamCountMax);

    bool longSymbols = (pInstance->m_log2HuffmanDistinctSymbolCountMax > 16);

    HuffmanGPUStreamInfo* dpStreamInfos = pInstance->getBuffer<HuffmanGPUStreamInfo>(streamCount);

    util::CudaScopedTimer timer(pInstance->Huffman.timerDecode);

    timer("Upload Info");

    // upload stream infos
    cudaSafeCall(hipMemcpyAsync(dpStreamInfos, pStreamInfos, sizeof(HuffmanGPUStreamInfo) * streamCount, hipMemcpyHostToDevice, pInstance->m_stream));
    // note: we don't sync on this upload - we trust that the caller won't overwrite/delete the array...

    timer("Decode");

    // get max number of symbols
    uint symbolCountPerStreamMax = 0;
    for(uint i = 0; i < streamCount; i++)
        symbolCountPerStreamMax = max(symbolCountPerStreamMax, pStreamInfos[i].symbolCount);

    if(symbolCountPerStreamMax == 0) {
        pInstance->releaseBuffer();
        return true;
    }

    // launch decode kernel
    uint threadCountPerStream = (symbolCountPerStreamMax + codingBlockSize - 1) / codingBlockSize;
    uint blockSize = min(192u, threadCountPerStream);
    blockSize = max(blockSize, HUFFMAN_LOOKUP_SIZE);
    assert(blockSize >= HUFFMAN_LOOKUP_SIZE);
    dim3 blockCount((threadCountPerStream + blockSize - 1) / blockSize, streamCount);

    if(longSymbols) {
        huffmanDecodeKernel<Symbol32><<<blockCount, blockSize, 0, pInstance->m_stream>>>(dpStreamInfos, codingBlockSize);
    } else {
        huffmanDecodeKernel<Symbol16><<<blockCount, blockSize, 0, pInstance->m_stream>>>(dpStreamInfos, codingBlockSize);
    }
    cudaCheckMsg("huffmanDecodeKernel execution failed");

    timer("Transpose");

    // launch transpose kernel
    dim3 blockSizeTranspose(TRANSPOSE_BLOCKDIM_X, TRANSPOSE_BLOCKDIM_Y);
    dim3 blockCountTranspose((symbolCountPerStreamMax + WARP_SIZE * codingBlockSize - 1) / (WARP_SIZE * codingBlockSize), streamCount);

    if(longSymbols) {
        switch(codingBlockSize) {
            case 32:
                huffmanDecodeTransposeKernel<Symbol32, 32><<<blockCountTranspose, blockSizeTranspose, 0, pInstance->m_stream>>>(dpStreamInfos);
                break;
            case 64:
                huffmanDecodeTransposeKernel<Symbol32, 64><<<blockCountTranspose, blockSizeTranspose, 0, pInstance->m_stream>>>(dpStreamInfos);
                break;
            case 128:
                huffmanDecodeTransposeKernel<Symbol32, 128><<<blockCountTranspose, blockSizeTranspose, 0, pInstance->m_stream>>>(dpStreamInfos);
                break;
            case 256:
                huffmanDecodeTransposeKernel<Symbol32, 256><<<blockCountTranspose, blockSizeTranspose, 0, pInstance->m_stream>>>(dpStreamInfos);
                break;
            default:
                assert(false);
        }
    } else {
        switch(codingBlockSize) {
            case 32:
                huffmanDecodeTransposeKernel<Symbol16, 32><<<blockCountTranspose, blockSizeTranspose, 0, pInstance->m_stream>>>(dpStreamInfos);
                break;
            case 64:
                huffmanDecodeTransposeKernel<Symbol16, 64><<<blockCountTranspose, blockSizeTranspose, 0, pInstance->m_stream>>>(dpStreamInfos);
                break;
            case 128:
                huffmanDecodeTransposeKernel<Symbol16, 128><<<blockCountTranspose, blockSizeTranspose, 0, pInstance->m_stream>>>(dpStreamInfos);
                break;
            case 256:
                huffmanDecodeTransposeKernel<Symbol16, 256><<<blockCountTranspose, blockSizeTranspose, 0, pInstance->m_stream>>>(dpStreamInfos);
                break;
            default:
                assert(false);
        }
    }
    cudaCheckMsg("huffmanDecodeTransposeKernel execution failed");

    timer();

    pInstance->releaseBuffer();

    return true;
}

}
